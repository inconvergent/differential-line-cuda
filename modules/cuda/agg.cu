#include "hip/hip_runtime.h"
#define THREADS _THREADS_

__global__ void agg(
  const int n,
  const int nz,
  const int zone_leap,
  const float *xy,
  int *zone_num,
  int *zone_node
){
  const int i = blockIdx.x*THREADS + threadIdx.x;

  if (i>=n){
    return;
  }

  const int ii = 2*i;
  const int zi = (int) floor(xy[ii]*nz);
  const int zj = (int) floor(xy[ii+1]*nz);
  const int z = zi*nz + zj;

  const int o = atomicAdd(&zone_num[z], 1);
  zone_node[z*zone_leap+o] = i;
}

