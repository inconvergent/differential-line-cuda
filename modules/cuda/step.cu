#include "hip/hip_runtime.h"
#define THREADS _THREADS_

__global__ void step(
  int n,
  int nz,
  int zone_leap,
  float *xy,
  float *dxy,
  float *tmp,
  float *link_len,
  float *link_curv,
  int *links,
  int *zone_num,
  int *zone_node,
  float stp,
  float reject_stp,
  float spring_stp,
  float near_rad,
  float far_rad
){
  const int i = blockIdx.x*THREADS + threadIdx.x;

  if (i>=n){
    return;
  }

  const int ii = 2*i;

  const int zi = (int)floor(xy[ii]*nz);
  const int zj = (int)floor(xy[ii+1]*nz);

  float sx = 0.0f;
  float sy = 0.0f;
  float dx = 0.0f;
  float dy = 0.0f;
  float dd = 0.0f;
  float force;


  int j;
  int jj;
  int zk;

  int cand_count = 0;
  int total_count = 0;

  tmp[i] = 0.0f;

  // unlinked
  for (int a=max(zi-1,0);a<min(zi+2,nz);a++){
    for (int b=max(zj-1,0);b<min(zj+2,nz);b++){
      zk = a*nz+b;
      for (int k=0;k<zone_num[zk];k++){

        j = zone_node[zk*zone_leap+k];

        if ((i==j) || (j == links[ii]) || (j == links[ii+1])){
          continue;
        }

        jj = 2*j;
        total_count += 1;
        dx = xy[ii] - xy[jj];
        dy = xy[ii+1] - xy[jj+1];
        dd = sqrt(dx*dx+dy*dy);

        if (dd<far_rad && dd>0.0f){
          cand_count += 1;
          force = (far_rad/dd-1.0);
          sx += force*dx*reject_stp;
          sy += force*dy*reject_stp;
        }
      }
    }
  }

  // linked
  for (int k=0;k<2;k++){
    j = links[ii+k];
    dx = xy[ii] - xy[2*j];
    dy = xy[ii+1] - xy[2*j+1];
    dd = sqrt(dx*dx + dy*dy);
    link_len[ii+k] = dd;
    if (dd>near_rad){
      sx -= dx/dd*spring_stp;
      sy -= dy/dd*spring_stp;
      tmp[i] = (float)dd;
    }
  }

  // curl


  float ax = xy[ii] - xy[2*links[ii]];
  float ay = xy[ii+1] - xy[2*links[ii]+1];
  dd = sqrt(ax*ax+ay*ay);
  ax/=dd;
  ay/=dd;

  float bx = xy[ii] - xy[2*links[ii+1]];
  float by = xy[ii+1] - xy[2*links[ii+1]+1];
  dd = sqrt(bx*bx+by*by);
  bx/=dd;
  by/=dd;

  link_curv[ii+1] = abs(ax*bx + ay*by);

  dxy[ii] = sx*stp;
  dxy[ii+1] = sy*stp;

  tmp[i] = (float)cand_count;

}

