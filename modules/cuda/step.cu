#include "hip/hip_runtime.h"
#define THREADS _THREADS_
#define PROX _PROX_

__global__ void step(
  int n,
  int nz,
  int zone_leap,
  float *xy,
  float *dxy,
  float *tmp,
  float *link_len,
  int *links,
  int *zone_num,
  int *zone_node,
  float stp,
  float reject_stp,
  float spring_stp,
  float near_rad,
  float far_rad
){
  const int i = blockIdx.x*THREADS + threadIdx.x;

  if (i>=n){
    return;
  }

  const int ii = 2*i;

  const int zi = (int)floor(xy[ii]*nz);
  const int zj = (int)floor(xy[ii+1]*nz);

  float sx = 0.0f;
  float sy = 0.0f;
  float dx = 0.0f;
  float dy = 0.0f;
  float dd = 0.0f;
  float force;


  int j;
  int jj;
  int zk;

  int cand_count = 0;
  int total_count = 0;

  int proximity[PROX];
  tmp[i] = 0.0f;

  for (int a=max(zi-1,0);a<min(zi+2,nz);a++){
    for (int b=max(zj-1,0);b<min(zj+2,nz);b++){
      zk = a*nz+b;
      for (int k=0;k<zone_num[zk];k++){

        j = zone_node[zk*zone_leap+k];

        if ((i==j) || (j == links[ii]) || (j == links[ii+1])){
          continue;
        }

        jj = 2*j;
        total_count += 1;
        dx = xy[ii] - xy[jj];
        dy = xy[ii+1] - xy[jj+1];
        dd = sqrt(dx*dx+dy*dy);

        if (dd<far_rad && dd>0.0f){
          proximity[cand_count] = j;
          cand_count += 1;
        }
      }
    }
  }

  for (int k=0;k<cand_count;k++){

    j = proximity[k];
    jj = 2*j;

    dx = xy[ii] - xy[jj];
    dy = xy[ii+1] - xy[jj+1];
    dd = sqrt(dx*dx + dy*dy);

    if (dd>0.0f){
      force = (far_rad/dd-1.0);
      sx += force*dx*reject_stp;
      sy += force*dy*reject_stp;
    }
  }

  for (int k=0;k<2;k++){
    j = links[ii+k];
    dx = xy[ii] - xy[2*j];
    dy = xy[ii+1] - xy[2*j+1];
    dd = sqrt(dx*dx + dy*dy);
    link_len[ii+k] = dd;
    if (dd>near_rad){
      sx -= dx/dd*spring_stp;
      sy -= dy/dd*spring_stp;
      tmp[i] = (float)dd;
    }
  }

  dxy[ii] = sx*stp;
  dxy[ii+1] = sy*stp;
  tmp[i] = (float)cand_count;

}

